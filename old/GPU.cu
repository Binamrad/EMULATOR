#include "hip/hip_runtime.h"
//This is an experimental implementation of the gpu emulator in CUDA
//it is much slower than the CPU emulator due to a significant overhead of lunching kernels
//Otherwise it works, for what few instructions it can process

extern "C" {
#include "GPU.h"
}

#include <stdint.h>
#include <string.h>

#define CORECOUNT 64
#define REGCOUNT 16//should we get 32?
#define CORESHIFT 4
#define MEMSIZE 1024*1024*8
#define MESSAGE_END 0
#define MESSAGE_NOTHING -1
#define MESSAGE_READ 1
#define MESSAGE_WRITE 2

typedef struct {
    unsigned long long clock;
    uint32_t data;
    uint32_t address;
    int status;
} gpu_message;

uint32_t *dev_registers;
uint8_t memory[1024*1024*8];//8 megabyte memory
uint8_t *dev_memory;//8 megabyte memory

gpu_message *message_area;
gpu_message *dev_message_area;


__device__ int getA(uint32_t inst) {
    inst >>= 20;
    return inst & 15;
}

__device__ int getB(uint32_t inst) {
    inst >>= 16;
    return inst & 15;
}

__device__ int getC(uint32_t inst) {
    return inst & 15;
}

__device__ int getIMM(uint32_t inst) {
    return inst & 0xffff;
}

extern "C"
void gpu_init() {
    memset(memory, 0, MEMSIZE);
    memory[256] = 0xff;
    hipMalloc(&dev_memory, MEMSIZE*sizeof(uint8_t));
    hipMalloc(&dev_registers, REGCOUNT*CORECOUNT*sizeof(uint32_t));
    hipMalloc(&dev_message_area, CORECOUNT*sizeof(gpu_message));
    message_area = (gpu_message*)malloc(CORECOUNT*sizeof(gpu_message));
    hipMemcpy(dev_memory,memory,MEMSIZE*sizeof(uint8_t),hipMemcpyHostToDevice);
}

//call after every branch
__device__ uint32_t gpu_getCurrentPC(uint32_t *PC) {
    uint32_t currentPC = 0xffffffff;    
    for(int i = 1; i < CORECOUNT; ++i) {
        if(PC[i] < currentPC) {
            currentPC = PC[i];
        }
    }
    return currentPC;
}

//saturating 31-bit addition of unsigned integers
//top bit of a and b is automatically ignored, no need to manually mask it out
__device__ uint32_t add31(uint32_t a, uint32_t b)
{
  uint32_t c = (a + b)&0x7fffffff;
  if (c<(a&0x7fffffff)) /* Can only happen due to overflow */
    c = 0x7fffffff;
  return c;
}

__device__ uint32_t fixpadd(uint32_t a, uint32_t b) {
    int64_t A = (int32_t)a;
    int64_t B = (int32_t)b;
    int64_t C = A+B;
    int32_t out = 0;
    if(C < 0) {
        out = 0x80000000;
        C = -C;
    }
    if(C > 0x7fffffff) {
        C = 0x7fffffff;
    }
    out += C;
    return out;
}

__global__ static void gpu_run_kernel(unsigned int program_counter_init, gpu_message *message_area, /*uint32_t *registers,*/ uint8_t *memory) {
    //__shared__ uint32_t PC[CORECOUNT];
    int myID = blockIdx.x;
    //PC[myID] = program_counter_init;
    unsigned int myPC = program_counter_init;
    int regOffset = (myID << CORESHIFT);    
    uint32_t currentPC = program_counter_init;
	uint32_t registers[REGCOUNT];
    unsigned long long cycles = 0;
    int running = 1;
    
    while(running) {
        uint32_t inst = *((uint32_t*)(memory+currentPC)); 
        int A = getA(inst);
        int B = getB(inst);
        int C = getC(inst);
        switch(inst) {
            case 0:
                cycles += 1;
                if(myPC == currentPC) {
                    myPC += 4;
                    registers[A+regOffset] = fixpadd(registers[B+regOffset], registers[C+regOffset]);
                }
                break;
            default:
                running = 0;
                cycles += 1;
                message_area[myID].status = 0;
                message_area[myID].clock = cycles;
                break;
        }
        //update PC
        currentPC += 4;
    }
}



extern "C"
unsigned long long gpu_run(unsigned int program_counter_init) {
    gpu_run_kernel<<<1,CORECOUNT>>>(program_counter_init, dev_message_area, /*dev_registers,*/ dev_memory);
    hipMemcpy(message_area,dev_message_area,CORECOUNT*sizeof(gpu_message),hipMemcpyDeviceToHost);
    //iterate through the messages, get highest clock count
    unsigned long long highestCount = 0;
    for(int i = 0; i < CORECOUNT; ++i) {
        if(message_area[i].clock > highestCount) highestCount = message_area[i].clock;
    }
    return highestCount;
}




